#include "hip/hip_runtime.h"
// System includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>



__global__ void kernel(float *g_nodes)
{
    // write data to global memory
    const unsigned int tid = threadIdx.x;
//    float nodes = g_nodes[tid];

    // use integer arithmetic to process all four bytes with one thread
    // this serializes the execution, but is the simplest solutions to avoid
    // bank conflicts for this very low number of threads
    // in general it is more efficient to process each byte by a separate thread,
    // to avoid bank conflicts the access pattern should be
    // g_data[4 * wtid + wid], where wtid is the thread id within the half warp
    // and wid is the warp id
    // see also the programming guide for a more in depth discussion.
//    g_data[tid] = ((((data <<  0) >> 24) - 10) << 24)
//                  | ((((data <<  8) >> 24) - 10) << 16)
//                  | ((((data << 16) >> 24) - 10) <<  8)
//                  | ((((data << 24) >> 24) - 10) <<  0);

	g_nodes[tid]=g_nodes[tid]*1.2;
}


extern "C" bool
runTest(const int argc, const char **argv, float *nodes, int node_count)
{
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    findCudaDevice(argc, (const char **)argv);

    const unsigned int num_threads = node_count*3;
//    assert(0 == (len % 4));
    const unsigned int mem_size = sizeof(float) * node_count*3;
//    const unsigned int mem_size_int2 = sizeof(int2) * len;

    // allocate device memory
    float *d_nodes;
    checkCudaErrors(hipMalloc((void **) &d_nodes, mem_size));
    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_nodes, nodes, mem_size,
                               hipMemcpyHostToDevice));
    // allocate device memory for int2 version
//    int2 *d_data_int2;
//    checkCudaErrors(hipMalloc((void **) &d_data_int2, mem_size_int2));
    // copy host memory to device
 //   checkCudaErrors(hipMemcpy(d_data_int2, data_int2, mem_size_int2,
  //                             hipMemcpyHostToDevice));

    // setup execution parameters
    dim3 grid(1, 1, 1);
    dim3 threads(num_threads, 1, 1);
    //dim3 threads2(len, 1, 1); // more threads needed fir separate int2 version
    // execute the kernel
    kernel<<< grid, threads >>>((float*) d_nodes);
    //kernel2<<< grid, threads2 >>>(d_data_int2);

    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    // compute reference solutions
//    char *reference = (char *) malloc(mem_size);
//    computeGold(reference, data, len);
//    int2 *reference2 = (int2 *) malloc(mem_size_int2);
//    computeGold2(reference2, data_int2, len);

    // copy results from device to host
    checkCudaErrors(hipMemcpy(nodes, d_nodes, mem_size,
                               hipMemcpyDeviceToHost));
//    checkCudaErrors(hipMemcpy(data_int2, d_data_int2, mem_size_int2,
//                               hipMemcpyDeviceToHost));

    // check result
    bool success = true;

//    for (unsigned int i = 0; i < len; i++)
//    {
//        if (reference[i] != data[i] ||
//            reference2[i].x != data_int2[i].x ||
//            reference2[i].y != data_int2[i].y)
//        {
//            success = false;
//        }
//    }

    // cleanup memory
    checkCudaErrors(hipFree(d_nodes));
//    checkCudaErrors(hipFree(d_data_int2));
//    free(reference);
//    free(reference2);

    return success;
}
