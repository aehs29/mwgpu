#include "hip/hip_runtime.h"
// System includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "cuPrintf.cu"



// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

__global__ void kern(int i)
{

i=i+i;
}

template <int BLOCK_SIZE> __global__ void kMatrixMult(float *A, float *B, float *C,int ARows, int ACols, int BRows, int BCols)
{

	int CRows=ARows;
	int CCols=BCols;
	
	int tx=threadIdx.x;
	int ty=threadIdx.y;


	float CValue = 0;

    int Row = blockIdx.y*BLOCK_SIZE + ty;
    int Col = blockIdx.x*BLOCK_SIZE + tx;

    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    for (int k = 0; k < (BLOCK_SIZE + ACols - 1)/BLOCK_SIZE; k++) {
		if (k*BLOCK_SIZE + tx < ACols && Row < ARows)   
			As[ty][tx] = A[Row*ACols + k*BLOCK_SIZE + tx];
		else                                              
			As[ty][tx] = 0.0;
		
		if (k*BLOCK_SIZE + ty < BRows && Col < BCols)  
			Bs[ty][tx] = B[(k*BLOCK_SIZE + ty)*BCols + Col];
		else
			Bs[ty][tx] = 0.0;
		
		__syncthreads();

		for (int n = 0; n < BLOCK_SIZE; ++n) CValue += As[ty][n] * Bs[n][tx];

		__syncthreads();
    }

    if (Row < CRows && Col < CCols) 
		C[((blockIdx.y * blockDim.y + ty)*CCols)+(blockIdx.x*blockDim.x)+tx]=CValue;



}

__global__ void kVectorAdd(float *A, float *B, float *C, unsigned int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

__global__ void kVectorScalar(float *A, const float scalar , int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        A[i] = A[i] * scalar;
    }
}


__global__ void kMatrixTranspose(float *A, float *B, int rows, int cols)
{

	int numElements=rows*cols;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
	// j = current row
	int j = i/cols;
		// k =  current col
	int k = i-(j*cols);

    if (i < numElements)
    {
		// A[j,k]=B[k,j]
		// A[k*rows+j]=B[j*rows+k]
		// if j=2, k =3
		// A[2,3]=12
		// B[3,2]=15
	    A[k*rows+j]=B[j*cols+k];
    }
}


__global__ void kMatVector(float *mat, float *vec, float *res, unsigned int RowsMat, unsigned int ColsMat) {
// Each thread computes one element of C
// by accumulating results into Cvalue
	float c = 0;
	//i is element in C to be computed
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if(i > RowsMat) return;

//    if (i < ColsA)
//    {
		for(int j=0;j<ColsMat;j++){
			c += mat[i*ColsMat+j]*vec[j];
		}
		res[i]=c;
//    }


}



//__global__ void kernel(float scale)
__global__ void kernel(float *g_nodes, float *u, float scale)
{
    // write data to global memory
    const unsigned int TID = threadIdx.x;
    const unsigned int BID = blockIdx.x;
	const unsigned int BDIM = blockDim.x;
//    float nodes = g_nodes[tid];

    // use integer arithmetic to process all four bytes with one thread
    // this serializes the execution, but is the simplest solutions to avoid
    // bank conflicts for this very low number of threads
    // in general it is more efficient to process each byte by a separate thread,
    // to avoid bank conflicts the access pattern should be
    // g_data[4 * wtid + wid], where wtid is the thread id within the half warp
    // and wid is the warp id
    // see also the programming guide for a more in depth discussion.
//    g_data[tid] = ((((data <<  0) >> 24) - 10) << 24)
//                  | ((((data <<  8) >> 24) - 10) << 16)
//                  | ((((data << 16) >> 24) - 10) <<  8)
//                  | ((((data << 24) >> 24) - 10) <<  0);

	// g_nodes[BDIM*BID+TID]=(g_nodes[BDIM*BID+TID])*scale;
	g_nodes[BDIM*BID+TID]=(g_nodes[BDIM*BID+TID]+u[BDIM*BID+TID])*scale;
//	scale*scale;
}
__global__ void kernelRemaining(float *g_nodes, float *u, float scale, unsigned int threadsDone)
{
    // write data to global memory
    const unsigned int TID = threadIdx.x;

	// g_nodes[threadsDone-1+TID]=(g_nodes[threadsDone-1+TID])*scale;
	g_nodes[threadsDone-1+TID]=(g_nodes[threadsDone-1+TID]+u[threadsDone-1+TID])*scale;
}

//float *cuda_data=NULL;
extern "C" void map_texture(void *cuda_dat, size_t siz,hipGraphicsResource *resource)
{
size_t size;
hipGraphicsResourceGetMappedPointer((void **)(&cuda_dat), &size, resource);
}

extern "C" bool
runTest(const int argc, const char **argv, float *buffer, float *u, int node_count, float scale)
{
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    //findCudaDevice(argc, (const char **)argv);



	// CUDA
//hipSetDevice(0);

//	cudaGLSetGLDevice(0);



const unsigned int maxThreadsBlock=512;
const unsigned int warpSize=32;
const unsigned int warpsBlock = maxThreadsBlock/warpSize;
const unsigned int totalThreads = node_count*3;
const unsigned int warps = totalThreads/warpSize;
const unsigned int mod = totalThreads%maxThreadsBlock;
unsigned int blocks = warps/warpsBlock;
unsigned int threadsDone = maxThreadsBlock*blocks;

unsigned int threadsX;

	if (totalThreads>512)
		threadsX=maxThreadsBlock;
	else{
		threadsX=totalThreads;
		blocks=1;
	}

    //const unsigned int num_threads = (node_count*3);
//	const unsigned int num_threads = 1;
//    assert(0 == (len % 4));
    const unsigned int mem_size = sizeof(float) * totalThreads;
//    const unsigned int mem_size_int2 = sizeof(int2) * len;

    // allocate device memory
    //float *d_nodes;
    //checkCudaErrors(hipMalloc((void **) &d_nodes, mem_size));
    // copy host memory to device
    //checkCudaErrors(hipMemcpy(d_nodes, nodes, mem_size,
    //                           hipMemcpyHostToDevice));
    // allocate device memory for int2 version
//    int2 *d_data_int2;
//    checkCudaErrors(hipMalloc((void **) &d_data_int2, mem_size_int2));
    // copy host memory to device
 //   checkCudaErrors(hipMemcpy(d_data_int2, data_int2, mem_size_int2,
  //                             hipMemcpyHostToDevice));

	//const int y=num_threads%512;

    hipError_t error;

	unsigned int mem_size_nodes = sizeof(float) * node_count*3;

	float *d_u;

	error = hipMalloc((void **) &d_u, mem_size_nodes);
	if (error != hipSuccess){
        printf("hipMalloc d_u returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
	error = hipMemcpy(d_u, u, mem_size_nodes, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_u,u) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }



    // setup execution parameters
    dim3 grid(blocks, 1, 1);
    dim3 threads(threadsX, 1, 1);
    //dim3 threads2(len, 1, 1); // more threads needed fir separate int2 version
//	printf("Executing Kernel, Threads: %u, Scale:%f\n",num_threads,scale);
    // execute the kernel
//    kernel<<< grid, threads >>>(scale);

/*
printf("Blocks:%u\n",blocks);
printf("ThreadsOnK:%u\n",threadsX);
printf("Remaining:%u\n",mod);
printf("threadsDone:%u\n", threadsDone);
*/
    kernel<<< grid, threads >>>(buffer, d_u, scale);
if(totalThreads>512)
	kernelRemaining<<< 1,mod >>>(buffer, d_u, scale, threadsDone);
    //kernel2<<< grid, threads2 >>>(d_data_int2);

	
    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    // compute reference solutions
//    char *reference = (char *) malloc(mem_size);
//    computeGold(reference, data, len);
//    int2 *reference2 = (int2 *) malloc(mem_size_int2);
//    computeGold2(reference2, data_int2, len);

    // copy results from device to host
   // checkCudaErrors(hipMemcpy(nodes, d_nodes, mem_size,
     //                          hipMemcpyDeviceToHost));
//    checkCudaErrors(hipMemcpy(data_int2, d_data_int2, mem_size_int2,
//                               hipMemcpyDeviceToHost));

    // check result
    bool success = true;

//    for (unsigned int i = 0; i < len; i++)
//    {
//        if (reference[i] != data[i] ||
//            reference2[i].x != data_int2[i].x ||
//            reference2[i].y != data_int2[i].y)
//        {
//            success = false;
//        }
//    }

    // cleanup memory
//    checkCudaErrors(hipFree(d_nodes));
//    checkCudaErrors(hipFree(d_data_int2));
//    free(reference);
//    free(reference2);

    return success;
}

extern "C" bool runTest2(const int argc, const char **argv, int i)
{
	hipSetDevice(0);
//	cudaGLSetGLDevice(0);
  dim3 grid(1, 1, 1);
    dim3 threads(2, 1, 1);

	  kern<<< grid, threads >>>(i);
    getLastCudaError("Kernel execution failed");

	  std::cout<<i;
	  return true;
}




extern "C" bool displacement (float *h_q, float *h_qo, float *h_qd, float *h_qdo, float *h_F, float *h_Fo, float *h_Ro, float *h_alpha, float * h_alphaI, float *h_beta, float *h_gama, float *h_eigenVecs, float h_h, float *h_u, unsigned int eigencount, unsigned int node_count, unsigned int node_dimensions, const int block_size){


/*

alpha,beta,gama: eigencount x eigencount

q,qd: eigencount x 1

F, u: (node_count x node_dimensions) x 1

R: (node_count x node_dimensions) x (node_count x node_dimensions)

Phi:  (node_count x node_dimensions) x eigencount

*/
	// u1, u2: eigenC x 1 - same as q

	// u3: (node_count x node_dimensions) x eigencount - same as Phi

	// u4: eigenC x 1 - same as q

	// u5: eigenC x 1 - same as q


/*#########---------Allocate variables---------------#########*/

	// Get size for pointers
	unsigned int size_nodes = node_count* node_dimensions;
	unsigned int size_eigen = eigencount;
	unsigned int mem_size_q = sizeof(float) * size_eigen;
	unsigned int mem_size_coef = sizeof(float) * (size_eigen*size_eigen);
	unsigned int mem_size_nodes = sizeof(float) * size_nodes;
	unsigned int mem_size_R = sizeof(float) * (size_nodes*size_nodes);
	unsigned int mem_size_Phi = sizeof(float) * (size_nodes*size_eigen);


	// Declare used variables on device
	float *d_alphaI, *d_qo, *d_beta, *d_qdo, *d_Ro, *d_Phi, *d_Fo, *d_gama, *d_alpha, *d_q, *d_qd;

	// Declare u's on device
	float *d_u1, *d_u2, *d_u3, *d_u3c, *d_u4, *d_u5, *d_u;

 // Error code to check return values for CUDA calls
    hipError_t error;

    error = hipMalloc((void **) &d_u1, mem_size_q);
	if (error != hipSuccess){
        printf("hipMalloc d_u1 returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMalloc((void **) &d_u2, mem_size_q);
	if (error != hipSuccess){
        printf("hipMalloc d_u2 returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMalloc((void **) &d_u3, mem_size_Phi);
	if (error != hipSuccess){
        printf("hipMalloc d_u3 returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
	error = hipMalloc((void **) &d_u3c, mem_size_Phi);
	if (error != hipSuccess){
        printf("hipMalloc d_u3c returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMalloc((void **) &d_u4, mem_size_q);
	if (error != hipSuccess){
        printf("hipMalloc d_u4 returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMalloc((void **) &d_u5, mem_size_q);
	if (error != hipSuccess){
        printf("hipMalloc d_u5 returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMalloc((void **) &d_alphaI, mem_size_coef);
	if (error != hipSuccess){
        printf("hipMalloc d_alphaI returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMalloc((void **) &d_qo, mem_size_q);
	if (error != hipSuccess){
        printf("hipMalloc d_qo returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMalloc((void **) &d_beta, mem_size_coef);
	if (error != hipSuccess){
        printf("hipMalloc d_beta returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_qdo, mem_size_q);
	if (error != hipSuccess){
        printf("hipMalloc d_qdo returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMalloc((void **) &d_Ro, mem_size_R);
	if (error != hipSuccess){
        printf("hipMalloc d_Ro returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_Phi, mem_size_Phi);
	if (error != hipSuccess){
        printf("hipMalloc d_Phi returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_Fo, mem_size_nodes);
	if (error != hipSuccess){
        printf("hipMalloc d_Fo returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_gama, mem_size_coef);
	if (error != hipSuccess){
        printf("hipMalloc d_gama returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_alpha, mem_size_coef);
	if (error != hipSuccess){
        printf("hipMalloc d_alpha returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_u, mem_size_nodes);
	if (error != hipSuccess){
        printf("hipMalloc d_u returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_qd, mem_size_q);
	if (error != hipSuccess){
        printf("hipMalloc d_qd returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
	}

    error = hipMalloc((void **) &d_q, mem_size_q);
	if (error != hipSuccess){
        printf("hipMalloc d_q returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
	}



// copy host memory to device

    error = hipMemcpy(d_alphaI, h_alphaI, mem_size_coef, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_alphaI,h_alphaI) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_qo, h_qo, mem_size_q, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_qo,h_qo) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_beta, h_beta, mem_size_coef, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_beta,h_beta) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_qdo, h_qdo, mem_size_q, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_qdo,h_qdo) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_Ro, h_Ro, mem_size_R, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_Ro,h_Ro) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_Phi, h_eigenVecs, mem_size_Phi, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_Phi,h_eigenVecs) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_Fo, h_Fo, mem_size_nodes, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_Fo,h_Fo) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_gama, h_gama, mem_size_coef, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_gama,h_gama) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_alpha, h_alpha, mem_size_coef, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_alpha,h_alpha) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

/*#########---------Ends variables allocation---------------#########*/



/*#########---------Get qd---------------#########*/

	// Setup Kernel parameters
/*###---------------------------------------------####*/

/*
MatByVec?
for(i=0;i<RowsA;i++)
for(j=0;j<ColsA;j++)
sum+=A[i*RowsA+j]*B[j];
 */

/*###----------------First Part--------------------####*/ 
        // First part
		// eigenC x 1
		// u1=(alpha-Identity)*qo       Can be done with VectorAdd Kernel, say whaat?

		//Launch one extra block to make it multiple of 32
	unsigned int threadsPB= 512;
	unsigned int BPG = (eigencount + threadsPB - 1) / threadsPB;
	dim3 threadsPerBlock(threadsPB);
	dim3 blocksPerGrid(BPG);
	printf("CUDA MatByVec (u1) kernel launch with %d blocks of %d threads\n", blocksPerGrid.x, threadsPerBlock.x);
		// Arguments: Matrix, Vector, Result, RowsMatrix, ColsMatrix
	kMatVector<<< blocksPerGrid, threadsPerBlock>>>(d_alphaI, d_qo, d_u1, eigencount, eigencount);
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to launch MatByVec kernel (u1) (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

	error = hipMemcpy(h_q, d_u1, mem_size_q, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_q,d_q) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
//	__syncthreads();
//--------------------Second Part-------------------	

	// Second part
	// eigenC x 1
	// u2=beta*qdo

	// Same kernel parameters as for u1
	printf("CUDA MatByVec (u2) kernel launch with %d blocks of %d threads\n", blocksPerGrid.x, threadsPerBlock.x);
		// Arguments: Matrix, Vector, Result, RowsMatrix, ColsMatrix
	kMatVector<<< blocksPerGrid, threadsPerBlock>>>(d_beta, d_qdo, d_u2, eigencount, eigencount);
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to launch MatByVec kernel (u2) (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

	error = hipMemcpy(h_q, d_u2, mem_size_q, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_q,d_q) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
//	__syncthreads();

//-------------------------Third Part .1 --------------------------	

	// Third part should be in a different variable to use it again on q

	// Third part .1
	// (node_count x node_dimensions) x eigencount
    // u3 = (Ro*Phi)

// 	unsigned int ColsB, RowsA;
// 	ColsB=eigencount;
// 	RowsA=size_nodes;

// 	// Different variables than MatByVec kernel
//     dim3 threads(block_size, block_size);
//     dim3 grid((ColsB+threads.x-1) / threads.x, (RowsA+threads.y-1) / threads.y);
// // Exec kernel
// 	kMatrixMult<16><<< grid, threads >>>(d_Ro, d_Phi, d_u3, size_nodes,size_nodes, size_nodes, size_eigen);
// 	    error = hipGetLastError();
//     if (error != hipSuccess)
//     {
//         fprintf(stderr, "Failed to launch matrixMult kernel (u3) (error code %s)!\n", hipGetErrorString(error));
//         exit(EXIT_FAILURE);
//     }

	// FOR NOW U3=Phi
	//d_u3=d_Phi;

//	__syncthreads()

//-------------------------Third Part .2-----------------------------

	// Third part .2
	// eigencount x (node_count x node_dimensions)
    // u3 = Transpose(u3)

	// Make a copy of u3 to transpose it
	error = hipMemcpy(d_u3c, d_Phi, mem_size_Phi, hipMemcpyDeviceToDevice);
    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_u3c,d_u3) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

	error = hipMemcpy(h_Ro, d_u3c, mem_size_Phi, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_q,d_q) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
	// New kernel parameters
	unsigned int numElements = size_nodes*size_eigen;
	blocksPerGrid=(numElements + threadsPB - 1) / threadsPB;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid.x, threadsPerBlock.x);
	// Arguments: Result, Original, # Elements in Matrix
    kMatrixTranspose<<< blocksPerGrid, threadsPerBlock>>>(d_u3,d_u3c, size_nodes, size_eigen);
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to launch matrixTranspose kernel (u3) (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

	error = hipMemcpy(h_Ro, d_u3, mem_size_Phi, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_q,d_q) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

//------------------------Third part .3-----------------------------
	// Third part .3
	// eigencount x 1
    // u4 = u3*Fo
	blocksPerGrid=(eigencount + threadsPB - 1) / threadsPB;
	printf("CUDA MatByVec (u4) kernel launch with %d blocks of %d threads\n", blocksPerGrid.x, threadsPerBlock.x);
		// Arguments: Matrix, Vector, Result, RowsMatrix, ColsMatrix
	kMatVector<<< blocksPerGrid, threadsPerBlock>>>(d_u3, d_Fo, d_u4, eigencount, size_nodes);
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to launch MatByVec kernel (u4) (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

	error = hipMemcpy(h_q, d_u4, mem_size_q, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_q,d_q) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
//-----------------------Part 4-----------------------
	// eigencount x 1
    // u5 = gama*u4

	// Same kernel params as u4
	printf("CUDA MatByVec (u5) kernel launch with %d blocks of %d threads\n", blocksPerGrid.x, threadsPerBlock.x);
		// Arguments: Matrix, Vector, Result, RowsMatrix, ColsMatrix
	kMatVector<<< blocksPerGrid, threadsPerBlock>>>(d_gama, d_u4, d_u5, eigencount, eigencount);
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to launch MatByVec kernel (u5) (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

	error = hipMemcpy(h_q, d_u5, mem_size_q, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_q,d_q) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

//--------------------------------------------
	// Fourht part sum all and divive by h
	// u2=u2+u5
	// vector + vector kernel u2=u2+u5

	// Same kernel parameters
	printf("CUDA MatByVec (u2.2) kernel launch with %d blocks of %d threads\n", blocksPerGrid.x, threadsPerBlock.x);
		// Arguments: VectorOrig&Res, Vector2, # of Elements
		// CHECK
		// Arguments: Vector1, Vector2, VectorResult,  # of Elements
	kVectorAdd<<< blocksPerGrid, threadsPerBlock>>>(d_u2, d_u5, d_u2, eigencount);
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to launch MatByVec kernel (u2.2) (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

	error = hipMemcpy(h_q, d_u2, mem_size_q, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_q,d_q) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
	// syncthreads()


	// qd=(u1+u2)/h

	// vector + vector u1=u1+u2
	// Same kernel parameters
	printf("CUDA VectorAdd (u1.2) kernel launch with %d blocks of %d threads\n", blocksPerGrid.x, threadsPerBlock.x);
		// Arguments: Vector1, Vector2, VectorResult,  # of Elements
	kVectorAdd<<< blocksPerGrid, threadsPerBlock>>>(d_u1, d_u2, d_qd, eigencount);
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to launch VectorAdd kernel (u1.2) (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	// vector by scalar kernel
	// qd=u1*1/h
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid.x, threadsPerBlock.x);
	// Arguments: vector, scalar, # elements
    kVectorScalar<<< blocksPerGrid, threadsPerBlock>>>(d_qd, 1/h_h,  eigencount);
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }


	error = hipMemcpy(h_q, d_qd, mem_size_q, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_q,d_q) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
/*#########---------Ends Get qd---------------#########*/


/*#########---------Get q---------------#########*/

	// First part
	// eigenC x 1
	// u1=alpha*qo       Can be done with VectorAdd Kernel, say whaat?

	// Same parameters as other kernels

	printf("CUDA MatByVec (u1.q) kernel launch with %d blocks of %d threads\n", blocksPerGrid.x, threadsPerBlock.x);
		// Arguments: Matrix, Vector, Result, RowsMatrix, ColsMatrix
	kMatVector<<< blocksPerGrid, threadsPerBlock>>>(d_alpha, d_qo, d_u1, eigencount, eigencount);
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to launch MatByVec kernel (u1.q) (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

	// Second part
	// eigenC x 1
	// q=u1+u2
// Same kernel parameters
	printf("CUDA VectorAdd (q) kernel launch with %d blocks of %d threads\n", blocksPerGrid.x, threadsPerBlock.x);
		// Arguments: Vector1, Vector2, VectorResult,  # of Elements
	kVectorAdd<<< blocksPerGrid, threadsPerBlock>>>(d_u1, d_u2, d_q, eigencount);
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to launch VectorAdd kernel (u1.2) (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	
/*#########---------Ends Get q---------------#########*/

/*#########---------Calculate u---------------#########*/

	//   (node_count x node_dimensions) x 1
	// u=Phi *q

// Same parameters as other kernels

	printf("CUDA MatByVec (u) kernel launch with %d blocks of %d threads\n", blocksPerGrid.x, threadsPerBlock.x);
		// Arguments: Matrix, Vector, Result, RowsMatrix, ColsMatrix
	kMatVector<<< blocksPerGrid, threadsPerBlock>>>(d_Phi, d_q, d_u, size_nodes, eigencount);
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to launch MatByVec kernel (u) (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

/*#########---------Ends Calculate u---------------#########*/


//Try not to delete U to be able to use it in the next kernel
	// Should I declare u  on device from main function?
	// May have to delete hipDeviceReset(); in other functions


/*#########---------Free Memory---------------#########*/
 // Copy result from device to host
    error = hipMemcpy(h_u, d_u, mem_size_nodes, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_u,d_u) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(h_q, d_q, mem_size_q, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_q,d_q) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(h_qd, d_qd, mem_size_q, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_qd,d_qd) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

	hipFree(d_u1);
	hipFree(d_u2);
	hipFree(d_u3);
	hipFree(d_u4);
	hipFree(d_u5);
	hipFree(d_alphaI);
	hipFree(d_qo);
	hipFree(d_beta);
	hipFree(d_qdo);
	hipFree(d_Ro);
	hipFree(d_Phi);
	hipFree(d_Fo);
	hipFree(d_gama);
	hipFree(d_alpha);
	hipFree(d_u);


/*#########---------Ends Free Memory---------------#########*/
	}
